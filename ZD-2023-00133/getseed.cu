#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>



__device__ int next(uint32_t state[32], int *ptr) {
    uint32_t next_state = state[(*ptr + 1) % 32] + state[(*ptr + 29) % 32];
    state[*ptr] = next_state;
    *ptr = (*ptr + 1) % 32;
    return next_state >> 1;
}

__device__ void init(uint32_t seed, uint32_t r[32]) {
    uint32_t state[344];
    state[0] = seed;
    for (int i = 1; i <= 30; i++)
        state[i] = (16807 * (int64_t)state[i - 1]) % 2147483647;
    for (int i = 31; i <= 33; i++)
        state[i] = state[i - 31];
    for (int i = 34; i < 344; i++)
        state[i] = (state[i - 3] + state[i - 31]);
    for (int i = 0; i < 31; i++)
        r[i] = state[313 + i];
}

__global__ void collide(uint8_t *ctx, uint32_t *ans, uint32_t *fnd,
                        uint32_t cnt, size_t start, size_t n_vars) {
    uint32_t bitrep = 0;
    bitrep |= cnt << 24;
    bitrep |= blockIdx.x << 16;
    bitrep |= threadIdx.x << 8;

    uint8_t *data = ctx;
    for (int lsb = 0; lsb < 256; lsb++) {
        uint32_t seed = bitrep | lsb;
        uint32_t r[32];
        init(seed, r);
        int ptr = 31;

        for (int i = 0; i < start; i++)
            next(r, &ptr);
        int valid = 1;
        for (int i = 0; i < n_vars; i++) {
            int res = next(r, &ptr) % 16;
            valid &= (res == data[i]);
        }
        if (valid) {
            *ans = seed;
            *fnd = 1;
            break;
        }
    }
}

uint8_t *read_from_file(const char *fname, size_t *n_vars, size_t *start) {
    FILE *fp = fopen(fname, "r");
    if (!fp) {
        printf("Failed to open %s\n", fname);
        exit(0);
    }
    fscanf(fp, "%zu%zu", n_vars, start);
    uint8_t *data = (uint8_t *)malloc(*n_vars);
    for (int i = 0; i < *n_vars; i++)
        fscanf(fp, "%hhd", &data[i]);
    fclose(fp);
    return data;
}

int validate(uint32_t seed, uint8_t *data, size_t n_vars, size_t start) {
    printf("Validating %u\n", seed);
    srandom(seed);
    for (int i = 0; i < start; i++)
        random();
    for (int i = 0; i < n_vars; i++) {
        if (random() % 16 != data[i])
            return 0;
    }
    return 1;
}

int run(uint8_t *host_ctx, size_t n_vars, size_t start) {
    uint8_t *device_ctx;
    hipMalloc((void **)&device_ctx, n_vars);
    hipMemcpy(device_ctx, host_ctx, n_vars, hipMemcpyHostToDevice);

    uint32_t *device_ans, *device_fnd, zero = 0;
    hipMalloc((void **)&device_ans, sizeof(uint32_t));
    hipMalloc((void **)&device_fnd, sizeof(uint32_t));
    hipMemcpy(device_ans, &zero, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(device_fnd, &zero, sizeof(uint32_t), hipMemcpyHostToDevice);

    for (int cnt = 0; cnt < 256; cnt++) {
        collide<<<256, 256>>>(device_ctx, device_ans, device_fnd, cnt, start, n_vars);
        uint32_t host_ans, host_fnd = 0;
        hipMemcpy(&host_ans, device_ans, sizeof(uint32_t), hipMemcpyDeviceToHost);
        hipMemcpy(&host_fnd, device_fnd, sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (host_fnd) {
            if (validate(host_ans, host_ctx, n_vars, start))
                printf("Seed is %u\n", host_ans);
            else
                printf("Validation failed.\n");
            return 1;
        }
    }
    return 0;
}

int main(int argc, char **argv) {
    if (argc == 0) exit(-1);
    if (argc == 1) {
        printf("Usage: %s input_file\n\n", argv[0]);
        printf("input_file:\n");
        printf("n start\n");
        printf("rng_0 rng_1 ... rng_n\n\n");
        printf("rng_0 is the first number after calling random() `start` times\n");
        exit(0);
    }

    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    for (int i = 0; i < num_devices; i++) {
        struct hipDeviceProp_t devInfo;
        hipGetDeviceProperties(&devInfo, i);
        printf("Device name: %s\n", devInfo.name);
    }

    size_t n_vars, start;
    uint8_t *host_ctx = read_from_file(argv[1], &n_vars, &start);

    run(host_ctx, n_vars, start);

    free(host_ctx);

    return 0;
}
